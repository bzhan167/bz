#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void histo_kernel(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins)
{
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
    extern __shared__ int histo_private[];
    for (int i =threadIdx;i < num_bins;i+= blockDim.x)
{
    histo_private[threadIdx.x] = 0;
}
    __syncthreads();
    // compute block's histogram
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < num_elements)
    {
        atomicAdd(&(histo_private[input[i]]), 1);
        i += stride;
    }
    // store to global histogram
    __syncthreads();
    while (threadIdx.x < num_bins)
{
    atomicAdd(&(histo[threadIdx.x]), histo_private[threadIdx.x]);
    threadIdx.x += blockDim.x;
}
    /*************************************************************************/
}
    void histogram(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) 
{
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 dim_grid(16, 1, 1);
    dim3 dim_block(512, 1, 1);
    histo_kernel<<<dim_grid, dim_block, num_bins*sizeof(unsigned int)>>>(input, bins, num_elements, num_bins);
    /*************************************************************************/
}
